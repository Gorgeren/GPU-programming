
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_vector_types.h>
#include <iostream>

#define CSC(call)                                            \
do {                                                         \
    hipError_t res = call;                                  \
    if(res != hipSuccess) {                                 \
        fprintf(stderr, "ERROR in %s: %d. Message: %s\n",    \
                __FILE__, __LINE__, hipGetErrorString(res));\
        exit(1);                                             \
    }                                                        \
} while(0);
__global__ void kernel(hipTextureObject_t tex, uchar4 *out, int w, int h) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int offsetx = blockDim.x * gridDim.x;
    int offsety = blockDim.y * gridDim.y;
    int x, y;
    uchar4 p;
    for(y = idy; y < h; y += offsety) {
        for(x = idx; x < w; x += offsetx) {
            p = tex2D<uchar4>(tex, x, y);
            int Y = static_cast<int>(0.299 * p.x + 0.587 * p.y + 0.114 * p.z);
            Y = Y > 255 ? 255 : Y;
            out[y * w + x] = make_uchar4(Y, Y, Y, p.w);
        }
    }
}
int main() {
    int w, h;
    std::cout << "enter filename\n";
    std::string in;
    std::cin >> in;
    FILE *fp = fopen(in.c_str(), "rb");
    fread(&w, sizeof(w), 1, fp);
    fread(&h, sizeof(h), 1, fp);
    uchar4 *data = new uchar4[w * h];
    fread(data, sizeof(uchar4), w * h, fp);
    fclose(fp);
    hipArray *arr;
    hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
    CSC(hipMallocArray(&arr, &ch, w, h));
    CSC(hipMemcpy2DToArray(arr, 0, 0, data, w * sizeof(uchar4), w * sizeof(uchar4), h, hipMemcpyHostToDevice));
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = arr;

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = false;

    hipTextureObject_t tex = 0;
    CSC(hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL));

    uchar4 *dev_out;
    CSC(hipMalloc(&dev_out, sizeof(uchar4)* w * h));
    kernel<<<dim3(16, 16), dim3(32, 32)>>> (tex, dev_out, w, h);
    CSC(hipGetLastError());
    CSC(hipMemcpy(data, dev_out, sizeof(uchar4) * w * h, hipMemcpyDeviceToHost));

    CSC(hipDestroyTextureObject(tex));
    CSC(hipFreeArray(arr));
    CSC(hipFree(dev_out));

    fp = fopen("out.data", "wb");
    fwrite(&w, sizeof(int), 1, fp);
    fwrite(&h, sizeof(int), 1, fp);
    fwrite(data, sizeof(uchar4), w * h, fp);
    fclose(fp);
    free(data);
    return 0;
}